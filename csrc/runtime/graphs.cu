#include "hip/hip_runtime.h"
#include "graphs.hpp"
#include "../common.hpp"

#ifdef TRELLIS_HAVE_CUDA
#include <hip/hip_runtime.h>
static __global__ void graph_dummy_kernel() {}

namespace trellis::graphs {

void demo_capture_and_launch() {
  hipStream_t stream = nullptr;
  hipStreamCreate(&stream);
  hipGraph_t graph = nullptr;
  hipGraphExec_t exec = nullptr;
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  graph_dummy_kernel<<<1,1,0,stream>>>();
  hipStreamEndCapture(stream, &graph);
  if (graph) {
    hipGraphInstantiate(&exec, graph, nullptr, nullptr, 0);
    hipGraphLaunch(exec, stream);
    hipStreamSynchronize(stream);
    hipGraphExecDestroy(exec);
    hipGraphDestroy(graph);
  }
  hipStreamDestroy(stream);
}

} // namespace trellis::graphs

#else

namespace trellis::graphs { void demo_capture_and_launch() {} }

#endif
