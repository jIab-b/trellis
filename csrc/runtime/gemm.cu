#include "gemm.hpp"
#include "../common.hpp"

#include <hipblas.h>
#include <mutex>
#include <hip/hip_fp16.h>

namespace trellis::gemm {

static hipblasHandle_t get_handle() {
  static hipblasHandle_t handle = nullptr;
  static std::once_flag once;
  std::call_once(once, [](){
    auto st = hipblasCreate(&handle);
    if (st != HIPBLAS_STATUS_SUCCESS) {
      TRELLIS_THROW("cuBLAS create failed");
    }
  });
  return handle;
}

static void check(hipblasStatus_t st, const char* what) {
  if (st != HIPBLAS_STATUS_SUCCESS) {
    TRELLIS_THROW(std::string("cuBLAS error in ") + what);
  }
}

void f32_rowmajor(const float* d_A, const float* d_B, float* d_C,
                  int m, int n, int k,
                  float alpha, float beta) {
  // Row-major to column-major mapping: see analysis.
  // We compute C^T = B^T @ A^T with column-major cuBLAS, dims n x m, k.
  // lda, ldb, ldc are leading dimensions for column-major inputs.
  hipblasHandle_t h = get_handle();
  hipblasOperation_t opA = HIPBLAS_OP_T; // A is row-major (m,k) -> treat as (k,m) column-major with transpose
  hipblasOperation_t opB = HIPBLAS_OP_T; // B is row-major (k,n) -> treat as (n,k) column-major with transpose

  int rows_Ct = n; // rows of C^T
  int cols_Ct = m; // cols of C^T
  int inner    = k; // shared dim

  const float* A_col = d_A;
  const float* B_col = d_B;
  float*       C_col = d_C;

  // In column-major, the leading dimension is the number of rows.
  int lda = m; // A^T has shape (k, m) but data from m,k row-major -> lda = m
  int ldb = k; // B^T has shape (n, k) but data from k,n row-major -> ldb = k
  int ldc = n; // C^T has shape (n, m)

  // Note: hipblasSgemm computes: C = alpha*op(A)*op(B) + beta*C with sizes (rows_Ct x cols_Ct)
  // hipblasSgemm arguments: (handle, opB, opA, rows_Ct, cols_Ct, inner, ...)
  // However, the conventional is hipblasSgemm(handle, opN, opN, m, n, k, ... A, lda, B, ldb, C, ldc)
  // We'll follow the standard order: C(n x m) = B(n x k)*A(k x m)
  check(
    hipblasSgemm(h,
                opB, opA,
                rows_Ct, cols_Ct, inner,
                &alpha,
                B_col, ldb,
                A_col, lda,
                &beta,
                C_col, ldc),
    "hipblasSgemm");
}

void f16_rowmajor_accum_f32(const void* d_A_half, const void* d_B_half, float* d_C,
                            int m, int n, int k,
                            float alpha, float beta) {
  hipblasHandle_t h = get_handle();
  // Map row-major to column-major via transpose, as above.
  hipblasOperation_t opA = HIPBLAS_OP_T;
  hipblasOperation_t opB = HIPBLAS_OP_T;
  int rows_Ct = n;
  int cols_Ct = m;
  int inner    = k;
  int lda = m; // A^T (k x m)
  int ldb = k; // B^T (n x k)
  int ldc = n; // C^T (n x m)
  check(
    hipblasGemmEx(h,
                 opB, opA,
                 rows_Ct, cols_Ct, inner,
                 &alpha,
                 d_B_half, HIP_R_16F, ldb,
                 d_A_half, HIP_R_16F, lda,
                 &beta,
                 d_C, HIP_R_32F, ldc,
                 HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT),
    "hipblasGemmEx f16->f32");
}

void f16_rowmajor_out_f16_accum_f32(const void* d_A_half, const void* d_B_half, void* d_C_half,
                                    int m, int n, int k,
                                    float alpha, float beta) {
  hipblasHandle_t h = get_handle();
  hipblasOperation_t opA = HIPBLAS_OP_T;
  hipblasOperation_t opB = HIPBLAS_OP_T;
  int rows_Ct = n;
  int cols_Ct = m;
  int inner    = k;
  int lda = m;
  int ldb = k;
  int ldc = n;
  check(
    hipblasGemmEx(h,
                 opB, opA,
                 rows_Ct, cols_Ct, inner,
                 &alpha,
                 d_B_half, HIP_R_16F, ldb,
                 d_A_half, HIP_R_16F, lda,
                 &beta,
                 d_C_half, HIP_R_16F, ldc,
                 HIPBLAS_COMPUTE_32F, HIPBLAS_GEMM_DEFAULT),
    "hipblasGemmEx f16->f16");
}

} // namespace trellis::gemm
